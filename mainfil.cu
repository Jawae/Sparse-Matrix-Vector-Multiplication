#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "utilities2.h"
#include "io.h"

using namespace std;

#define BIN_MAX 5
#define ROW_MAX 1024
#define THREAD_LOAD 5

__inline__ __device__
float warpReduceSum(float val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2)
    	val += __shfl_down(val, offset);
  return val;
}

__inline__ __device__
float blockReduceSum(float val) {

  static __shared__ int shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / 32.0) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}

__global__ void spmv(float * __restrict__ values, int * __restrict__ col_idx, int * __restrict__ row_off,float * __restrict__ vect,\
 float res[], int  m, int  n, int *  bin, int  bin_size,int  N, int nnz)
{
	int tid = threadIdx.x;
	int lid = tid%32;
	int vid = tid/32;
	float sum = 0;
	int row = bin[blockIdx.x];
	int row_idx = row_off[row];
	int next_row_idx;
	if(row < (m-1))
		next_row_idx = row_off[row+1];
	else
		next_row_idx = nnz;
	for(int i = row_idx + tid; i < next_row_idx; i+= 1<<(N-1))
	{
		sum += values[i] * vect[col_idx[i]];
	}

	__syncthreads();

	sum = blockReduceSum(sum);



	if(lid == 0 && vid == 0)
		res[row] = sum;

}

__global__ void row_specific_spmv(float* values,int* col_idx,int* row_off, float* x,float res[],int m,int n,int nnz,int row,int noOfThreads){
  int tid = threadIdx.x;
  int lid = tid%32;
  int vid = tid/32;

  float sum = 0;

  int row_idx = row_off[row];
	int next_row_idx;
	if(row < (m-1))
		next_row_idx = row_off[row+1];
	else
		next_row_idx = nnz;
	for(int i = row_idx + tid; i < next_row_idx; i+= noOfThreads)
	{
		sum += values[i] * x[col_idx[i]];
	}

	__syncthreads();

	sum = blockReduceSum(sum);

	if(lid == 0 && vid == 0)
		res[row] = sum;

}
__global__ void dynamicParallelParent(float * values, int * col_idx, int * row_off,float * x,\
 float res[], int  m, int  n, int nnz,int*  G1, int G1_size){
  int tid = threadIdx.x;
  // printf("threadIdx = %d\n",tid);
  int row = G1[tid];
  int row_idx = row_off[row];
  int next_row_idx;
  if(row==m-1){
    next_row_idx = nnz;
  }
  else
    next_row_idx = row_off[row+1];

  int NNZ = next_row_idx - row_idx;
  int bsize = (NNZ-1)/THREAD_LOAD + 1;

  row_specific_spmv<<<1,bsize>>>(values,col_idx,row_off,x,res,m,n,nnz,row,bsize);
 }
int calc_bin_index(int nnz){
  if(nnz==0 | nnz==1)
    return nnz;

  int cnt = 0,orig_nnz = nnz;
  while(nnz>0){
    nnz>>=1;
    cnt++;
  }
  if(!(orig_nnz & (orig_nnz-1)))
    return cnt-1;
  else return cnt;

}
// Matrix : m x n
// Vector : n x 1
float* driver(float *values, int *col_idx, int* row_off, float* x, float* y, int m, int n, int nnz){
  int max_nnz = INT_MIN;
  for(int i=1;i<m;i++)
    max_nnz = max(max_nnz,row_off[i]-row_off[i-1]);
  max_nnz = max(max_nnz, m-row_off[m-1]);
  // cout<<"max_nnz = "<<max_nnz<<"\n";

//Timer setup
  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int max_bins = calc_bin_index(max_nnz);
   cout<<"max_bins = "<<max_bins<<"\n";
  vector<int> bins[max_bins+1];

  for(int i = 1;i<m;i++){
    int nnz = row_off[i]-row_off[i-1];
    int bin_index = calc_bin_index(nnz);
    bins[bin_index].push_back(i-1);
  }

  int last_nnz = nnz-row_off[m-1];
  bins[calc_bin_index(last_nnz)].push_back(m-1);

  for(int i=0;i<=max_bins;i++){
    cout<<i<<"-->"<<bins[i].size();
    cout<<"\n";
  }

  int *dcol_idx, *drow_off;
  float *dvect, *dres, *dvalues;

//Memory Allocation
  cout<<"Allocating memory\n";
  hipEventRecord(start);
  hipMalloc((void**)&dcol_idx, (nnz)*sizeof(int));
  hipMalloc((void**)&drow_off, (m)*sizeof(int));
  hipMalloc((void**)&dvect, (n)*sizeof(float));
  hipMalloc((void**)&dres, (m)*sizeof(float));
  hipMalloc((void**)&dvalues, (nnz)*sizeof(float));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  cout<<"Memory Allocation successful: "<<milliseconds<<"ms\n";


  //Copying memory to GPU
  cout<<"Copying memory to GPU\n";
  hipEventRecord(start);
  hipMemcpy(dcol_idx, col_idx, (nnz)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(drow_off, row_off, (m)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dvect, x, (n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dvalues, values, (nnz)*sizeof(float), hipMemcpyHostToDevice);
  hipMemset(dres, 0, n * sizeof(float));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  cout<<"Memory copy complete: "<<milliseconds<<"ms\n";

  float kernel_time = 0;
  //Calculate G2
  for(int i = 1; i <=min(max_bins,BIN_MAX); i++)
	{
		if(bins[i].size()>0)
		{
      cout<<"Currently Bin "<<i<<endl;
      int *dbin;
			hipMalloc((void**)&dbin, bins[i].size() * sizeof(int));

      int arr[bins[i].size()]; //Temporary array to store a single bin
			for(int j = 0; j < bins[i].size();j++)
				arr[j] = bins[i][j];

			hipMemcpy(dbin, arr, (bins[i].size())*sizeof(int), hipMemcpyHostToDevice);

			int dimBlock = (1 << (i - 1));
      if(dimBlock > 1024)
        dimBlock = 1024;
			dim3 dimGrid(bins[i].size());
      cout<<"Total No of threads: "<<dimBlock*bins[i].size()<<endl;

      cout<<"Executing Kernel: ";
      hipEventRecord(start);
			spmv<<<dimGrid,dimBlock>>>(dvalues, dcol_idx, drow_off, dvect, dres, m, n, dbin, bins[i].size(), i, nnz);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      cout<<"Bin "<<i<<" execution complete: "<<milliseconds<<"ms\n";

			hipFree(dbin);

      kernel_time += milliseconds;
		}
	}

  printf("\n\nGPU time taken for G2: %f\n\n", kernel_time);

  int *G1,*dG1;
  G1 = (int*)malloc(sizeof(int)*(m));
  int no_of_bigrows = 0;
  for(int i = BIN_MAX+1;i<=max_bins;i++){
    for(int j=0;j<bins[i].size();j++){
      G1[no_of_bigrows++] = bins[i][j];
    }
  }

  cout<<"no_of_bigrows = "<<no_of_bigrows<<"\n";
  //cout<<"Big rows = \n";
  //for(int i=0;i<no_of_bigrows;i++) cout<<G1[i]<<" ";
  cout<<"\n\n";
  hipMalloc((void**)&dG1,(no_of_bigrows)*sizeof(int));
  hipMemcpy(dG1,G1,no_of_bigrows*sizeof(int),hipMemcpyHostToDevice);

  cout<<"Executing G1 Kernel: ";
  hipEventRecord(start);
  dynamicParallelParent<<<1,no_of_bigrows>>>(dvalues, dcol_idx, drow_off, dvect, dres, m, n, nnz, dG1, no_of_bigrows);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  cout<<"Time taken for G1: "<<milliseconds<<" ms\n";

  float* kres = (float*)malloc(m*sizeof(float));
  hipMemcpy(kres, dres, (m)*sizeof(float), hipMemcpyDeviceToHost);

  //cout<<"result = ";
  //for(int i=0;i<m;i++) cout<<kres[i]<<" ";
  //cout<<"\n";
  return kres;
}

int main(){
   int n,m,nnz=0;
   int nnz_max;
   float *x;
   srand (time(NULL)); //Set current time as random seed.
  // cout<<"m,n = ";
  // cin>>m>>n;
  //
  // cout<<"matrix = \n";
  // vector<vector<float> > mat(m,vector<float>(n));
  // for(int i=0;i<m;i++){
  //   for(int j=0;j<n;j++){
  //     cin>>mat[i][j];
  //     if(mat[i][j]!=0){
  //       nnz++;
  //     }
  //   }
  // }
  // float* values = (float*)calloc(1,nnz*sizeof(float));
  // int* row_off = (int*)calloc(1,m*sizeof(int));
  // int* col_idx = (int*)calloc(1,nnz*sizeof(int));
  // to_csr(mat,values,col_idx,row_off,m,n);
  //
  //
  // cout<<"x = ";
  // float* x = (float*) malloc(n*sizeof(float));
  // for(int i=0;i<n;i++) cin>>x[i];
  //
  // //cout<<"y = ";

  // //for(int i=0;i<m;i++) cin>>y[i];
  conv(nnz, m, n, nnz_max);
  x = vect_gen(n);
  float* y = (float*) malloc(m*sizeof(float));
  float *res = new float[m];

  clock_t begin = clock();
  simple_spmv(res, x, values, col_idx, row_off, nnz, m, n);
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  cout<<"\nTime taken for sequential: "<<elapsed_secs*1000<<"\n\n\n";




   y = driver(values,col_idx,row_off,x,y,m,n,nnz);
   checker(y,res,m);


}
