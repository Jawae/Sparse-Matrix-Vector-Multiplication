#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "utilities.h"

using namespace std;



__global__ void spmv(float *values, int *col_idx, int *row_off,float * vect, float res[], int m, int n, int *bin, int bin_size,int bin_row_len)
{
	int tid = threadIdx.x;
	int lid = tid%32;
	int vid = tid/32;
	float sum = 0;
	int row = bin[lid];
	int row_idx = row_off[row];
	int next_row_idx = row_off[row+1];

	for(int i = row_idx + vid; i < next_row_idx; i+= 1<<(bin_row_len - 1))
	{
		sum += values[i] * vect[col_idx[i]];
	} 

	for(int i = bin_size; i > 0; i--)
		sum += __shfl_down(sum,i);

	//printf("sum = ");

	if(vid == 0)
		res[row] += sum;
	
}

int main()
{
	srand (time(NULL));
	int m = 5, n = 5;
	int nnz = 0, nnz_row[m], nnz_max = 0; 
	float *mat[m], *vect, *res;
	float *values;
	int *col_idx, *row_off;



	for(int i = 0; i < m; i++)
	{
		mat[i] = sparse_gen(n, nnz, nnz_row[i], nnz_max);
	}

	cout<<"\nMatrix generated: \n";
	display_matrix(mat, m , n);

	vect = vect_gen(n);
	cout<<"\nVector generated: \n";
	display_vector(vect, n);

	cout<<"NNZ: "<<nnz<<endl;

	values = new float [nnz];
	col_idx = new int[nnz];
	row_off = new int[m];

	vector <vector<int> > bins(nnz_max+1);


	to_csr(mat, values, col_idx, row_off, m, n);
	display_csr(values, col_idx, row_off, nnz, m);


	res = new float[n];

	simple_spmv(res, vect, values, col_idx, row_off, nnz, m, n);

	cout<<"Result vector: \n";
	display_vector(res, n);

	calculate_bin_size(bins, nnz_row, m);

	for (int i = 0; i < nnz_max+1; ++i)
	{

		cout<<"Bin Size: "<<i<<endl;
		for(int j = 0; j < bins[i].size(); j++)
		{
			cout<<bins[i][j]<<" ";
		}

		cout<<endl;
	}



	// CUDA stuff

	int *dcol_idx, *drow_off, *dbin;
	float *dvect, *dres, *dvalues;
	float milliseconds = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	cout<<"Allocating memory\n";
	hipEventRecord(start);
	hipMalloc((void**)&dcol_idx, (nnz)*sizeof(int));
	hipMalloc((void**)&drow_off, (m)*sizeof(int));
	hipMalloc((void**)&dvect, (n)*sizeof(float));
	hipMalloc((void**)&dres, (n)*sizeof(float));
	hipMalloc((void**)&dvalues, (nnz)*sizeof(float));
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"Memory Allocation successful: "<<milliseconds<<"ms\n";

	cout<<"Copying memory\n";
	hipEventRecord(start);
	hipMemcpy(dcol_idx, col_idx, (nnz)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(drow_off, row_off, (m)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dvect, vect, (n)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dvalues, values, (nnz)*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(dres, 0, n * sizeof(float));
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"Memory copy complete: "<<milliseconds<<"ms\n";

	for(int i = 1; i < bins.size(); i++)
	{
		if(bins[i].size()>0)
		{
			cout<<"Currently Bin "<<i<<endl;
			hipMalloc((void**)&dbin, bins[i].size() * sizeof(int));

			int arr[bins[i].size()];
			for(int j = 0; j < bins[i].size();j++)
				arr[j] = bins[i][j];

			hipMemcpy(dbin, arr, (bins[i].size())*sizeof(int), hipMemcpyHostToDevice);

			int dimBlock = (1 << (i - 1)) * bins[i].size() ;
			cout<<"No of threads: "<<dimBlock<<endl;
			//dim3 dimGrid(bins[i].size());
			cout<<"Executing Kernel: ";
			hipEventRecord(start);
			spmv<<<1,dimBlock>>>(dvalues, dcol_idx, drow_off, dvect, dres, m, n, dbin, bins[i].size(), i);
			hipEventRecord(stop);

			hipEventSynchronize(stop);
			hipEventElapsedTime(&milliseconds, start, stop);
			cout<<"Bin "<<i<<" execution complete: "<<milliseconds<<"ms\n";
			hipFree(dbin);
		}
		
	}

	hipEventRecord(start);
	hipMemcpy(res, dres, (n)*sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	cout<<"Output Vector: ";

	display_vector(res,n);

	cout<<"Freeing memory\n";
	hipEventRecord(start);
	hipFree(dcol_idx);
	hipFree(drow_off);
	hipFree(dvect);
	hipFree(dres);
	hipFree(dvalues);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<"Memory Freed: "<<milliseconds<<"ms\n";

}